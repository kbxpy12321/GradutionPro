#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>  // stdio functions are used since C++ streams aren't necessarily thread safe
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
#include <helper_functions.h>
#include <Matrix.h>
#include <MatrixCalculation.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_new.h>
#include <thrust/device_ptr.h>

using namespace std;
__global__ void kernelAddConstant(int *g_a, const int b)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	g_a[idx] += b;
}

__global__ void kernelMatrixMul(int *matrixA, int *matrixB, int *matrixC, int sameside) {
	
	int col = sizeof(matrixB) / sizeof(matrixB[0]) / sameside;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	matrixC[idx] = 0;
	for (int i = 0; i < sameside; i++) {
		matrixC[idx] += matrixA[blockIdx.x * sameside + i] * matrixB[i * col + threadIdx.x];
	}
}

__global__ void kernelMatrixAdd(Matrix *matrixA, Matrix *matrixB) {
	
}

extern "C" Matrix *matrixMulByCuda(Matrix *matrixA, Matrix* matrixB) { 
	if (matrixA == NULL || matrixB == NULL || matrixA->getType() != INTEGER || matrixB->getType() != INTEGER || matrixB->getCol() > 1024) {
		return nullptr;
	}
	int row = matrixA->getRow();
	int col = matrixB->getCol();
	int sameSide = matrixA->getCol();
	int fullLen = matrixA->getRow() * matrixB->getCol();
	thrust::device_vector<int> res(fullLen);
	thrust::device_vector<int> matrixATmp(matrixA->returnVector());
	thrust::device_vector<int> matrixBTmp(matrixB->returnVector());
	int* tmpA = thrust::raw_pointer_cast(matrixATmp.data());
	int* tmpB = thrust::raw_pointer_cast(matrixATmp.data());
	int* tmpC = thrust::raw_pointer_cast(res.data());
	kernelMatrixMul << <row, col>> > (tmpA, tmpB, tmpC, sameSide);
	//matrixA->printMatrix();
	return nullptr;
}

extern "C" int testInCuda() {
	int num_gpus = 0;
	hipGetDeviceCount(&num_gpus);
	return num_gpus;
}

extern "C" int testCudaOpenMP() {
	int num_gpus = 0;   // number of CUDA GPUs

	//printf("%s Starting...\n\n", argv[0]);

	/////////////////////////////////////////////////////////////////
	// determine the number of CUDA capable GPUs
	//
	hipGetDeviceCount(&num_gpus);

	if (num_gpus < 1)
	{
		printf("no CUDA capable devices were detected\n");
		return 1;
	}

	/////////////////////////////////////////////////////////////////
	// display CPU and GPU configuration
	//
	printf("number of host CPUs:\t%d\n", omp_get_num_procs());
	printf("number of CUDA devices:\t%d\n", num_gpus);

	for (int i = 0; i < num_gpus; i++)
	{
		hipDeviceProp_t dprop;
		hipGetDeviceProperties(&dprop, i);
		printf("   %d: %s\n", i, dprop.name);
	}

	printf("---------------------------\n");


	/////////////////////////////////////////////////////////////////
	// initialize data
	//
	unsigned int n = num_gpus * 8192;
	unsigned int nbytes = n * sizeof(int);
	int *a = 0;     // pointer to data on the CPU
	int b = 3;      // value by which the array is incremented
	a = (int *)malloc(nbytes);

	if (0 == a)
	{
		printf("couldn't allocate CPU memory\n");
		return 1;
	}

	for (unsigned int i = 0; i < n; i++)
		a[i] = i;


	////////////////////////////////////////////////////////////////
	// run as many CPU threads as there are CUDA devices
	//   each CPU thread controls a different device, processing its
	//   portion of the data.  It's possible to use more CPU threads
	//   than there are CUDA devices, in which case several CPU
	//   threads will be allocating resources and launching kernels
	//   on the same device.  For example, try omp_set_num_threads(2*num_gpus);
	//   Recall that all variables declared inside an "omp parallel" scope are
	//   local to each CPU thread
	//
	omp_set_num_threads(num_gpus);  // create as many CPU threads as there are CUDA devices
	//omp_set_num_threads(2*num_gpus);// create twice as many CPU threads as there are CUDA devices
#pragma omp parallel
	{
		unsigned int cpu_thread_id = omp_get_thread_num();
		unsigned int num_cpu_threads = omp_get_num_threads();

		// set and check the CUDA device for this CPU thread
		int gpu_id = -1;
		checkCudaErrors(hipSetDevice(cpu_thread_id % num_gpus));   // "% num_gpus" allows more CPU threads than GPU devices
		checkCudaErrors(hipGetDevice(&gpu_id));
		printf("CPU thread %d (of %d) uses CUDA device %d\n", cpu_thread_id, num_cpu_threads, gpu_id);

		int *d_a = 0;   // pointer to memory on the device associated with this CPU thread
		int *sub_a = a + cpu_thread_id * n / num_cpu_threads;   // pointer to this CPU thread's portion of data
		unsigned int nbytes_per_kernel = nbytes / num_cpu_threads;
		dim3 gpu_threads(128);  // 128 threads per block
		dim3 gpu_blocks(n / (gpu_threads.x * num_cpu_threads));

		checkCudaErrors(hipMalloc((void **)&d_a, nbytes_per_kernel));
		checkCudaErrors(hipMemset(d_a, 0, nbytes_per_kernel));
		checkCudaErrors(hipMemcpy(d_a, sub_a, nbytes_per_kernel, hipMemcpyHostToDevice));
		kernelAddConstant << <gpu_blocks, gpu_threads >> > (d_a, b);

		checkCudaErrors(hipMemcpy(sub_a, d_a, nbytes_per_kernel, hipMemcpyDeviceToHost));
		checkCudaErrors(hipFree(d_a));

	}
	printf("---------------------------\n");

	if (hipSuccess != hipGetLastError())
		printf("%s\n", hipGetErrorString(hipGetLastError()));


	////////////////////////////////////////////////////////////////
	// check the result
	//
	//bool bResult = correctResult(a, n, b);

	if (a)
		free(a); // free CPU memory

	//exit(bResult ? EXIT_SUCCESS : EXIT_FAILURE);

	return 1;
}

